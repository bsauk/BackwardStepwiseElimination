#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
extern "C" {
#include "subset.h"
}

int fact(int z) {
  
  int f = 1;
  if(z == 0) {
    return f;
  } else {
    for(int i=1; i <= z; i++) {
      f = f*i;
    }
  } 
  return (f);
}

unsigned long comb(int n, int r) {
  unsigned long f = 1;
  if(n==r) {
    return f;
  } else {
    for(int i=n; i>n-r; i--) {
      f = f*i;
    }
  }

  f = f/fact(r);

  return f;
}  

void matPrint(double *A, int m, int count) {
  char p[4]= {'A', 'B', 'x', '\0'};
  for(int i=0; i<m; i++) {
    printf("%c(%d)=%f;\n", p[count], i+1, A[i]);
  }
}

void matrixInit(int in, double **A, double **B, int batchSize, int lim1, int lim2) {
    
  // Code to either make matrix or initialize it
  int i=0; 
  int j=0; 
  double div = RAND_MAX/1000;
  srand((unsigned) in);
  for(j=0; j<batchSize; j++) {
    for(i=0; i<lim1; i++) { 
      A[j][i] = rand()/div;
    }
  }
  for(j=0; j<batchSize; j++) {
    for(i=0; i<lim2; i++) {
      B[j][i] = rand()/div;
    }
  }
}

//float testMagma(int m, int n, int r, int batchSize) {


//}


float testSubset(int m, int n, int r, int batchSize) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipblasHandle_t cublas_handle;

  hipblasCreate(&cublas_handle);

  int info, i, nrhs, lda;
  float gpuTime = 0, iterTime = 0; 
  
  info = 0;
  nrhs = 1;
  lda = ((m+15)/16)*16;
  int *devInfoArray;
  double **A, **B;
  double **dA, **dB, **hdA, **hdB;

  hipMalloc((void**)&devInfoArray, batchSize*sizeof(int));
  A = (double **)malloc(batchSize*sizeof(double*));
  B = (double **)malloc(batchSize*sizeof(double*));
  for(i=0; i<batchSize; i++) {
    A[i] = (double *)malloc(lda*r*sizeof(double));
    B[i] = (double *)malloc(lda*sizeof(double));
  }

  hdA = (double **)malloc(batchSize*sizeof(double*));
  hdB = (double **)malloc(batchSize*sizeof(double*));

  for(i=0; i<batchSize; i++) {
    hipMalloc((void**)&hdA[i], lda*r*sizeof(double));
    hipMalloc((void**)&hdB[i], lda*sizeof(double));
  }
  
  hipMalloc((void**)&dA, batchSize*sizeof(double*));
  hipMalloc((void**)&dB, batchSize*sizeof(double*));

  matrixInit(0, A, B, batchSize, lda*r, lda);

  hipEventRecord(start);
  // Code to calculate SSE from subset
  for(i=0; i<batchSize; i++) {
    hipblasSetMatrix(m, r, sizeof(double), A[i], lda, hdA[i], lda);
    hipblasSetMatrix(m, 1, sizeof(double), B[i], lda, hdB[i], lda);
  }
  hipMemcpy(dA, hdA, batchSize*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(dB, hdB, batchSize*sizeof(double*), hipMemcpyHostToDevice);

  hipblasDgelsBatched(cublas_handle, HIPBLAS_OP_N, m, r, nrhs, dA, lda, dB, lda, &info, devInfoArray, batchSize);
    
  hipDeviceSynchronize();

  for(i=0; i<batchSize; i++) {
    hipblasGetMatrix(m, 1, sizeof(double), hdB[i], lda, B[i], lda);
  } 

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  iterTime = 0;
  hipEventElapsedTime(&iterTime, start, stop);

  gpuTime = gpuTime + iterTime;

  for(int i=0; i<batchSize; i++) {
    free(A[i]);
    free(B[i]);
    hipFree(hdA[i]);
    hipFree(hdB[i]);
  }
  if(dA) hipFree(dA);
  if(dB) hipFree(dB);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipblasDestroy(cublas_handle);
  if(A) free(A);
  if(B) free(B);
  if(hdA) free(hdA);
  if(hdB) free(hdB);

  return gpuTime;
}


float allSubset(int *bestSubset, double *bestSSE, int m, int n, int r, unsigned long batchSize) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipblasHandle_t cublas_handle;

  hipblasCreate(&cublas_handle);

  int info, i, count, nrhs, lda;
  float gpuTime = 0, iterTime = 0; 
  
  count = 1;
  info = 0;
  nrhs = 1;
  lda = ((m+15)/16)*16;
  int *devInfoArray;
  double **A, **B;
  double **dA, **dB, **hdA, **hdB;

  unsigned long bigNum = 1.25e9 / (lda*(r+1)*sizeof(double));
  while(batchSize > bigNum) {
    batchSize = (batchSize+1)/2; 
    count = count+1;
  }

  hipMalloc((void**)&devInfoArray, batchSize*sizeof(int));
  A = (double **)malloc(batchSize*sizeof(double*));
  B = (double **)malloc(batchSize*sizeof(double*));
  for(i=0; i<batchSize; i++) {
    A[i] = (double *)malloc(lda*r*sizeof(double));
    B[i] = (double *)malloc(lda*sizeof(double));
  }

  hdA = (double **)malloc(batchSize*sizeof(double*));
  hdB = (double **)malloc(batchSize*sizeof(double*));

  for(i=0; i<batchSize; i++) {
    hipMalloc((void**)&hdA[i], lda*r*sizeof(double));
    hipMalloc((void**)&hdB[i], lda*sizeof(double));
  }
  
  hipMalloc((void**)&dA, batchSize*sizeof(double*));
  hipMalloc((void**)&dB, batchSize*sizeof(double*));

  /* CODE TO READ IN OR WRITE A MATRIX */
  for(int block=0; block<count; block++) {
    matrixInit(block, A, B, batchSize, lda*r, lda);
    hipEventRecord(start);
    // Code to calculate SSE from subset
    for(i=0; i<batchSize; i++) {
      hipblasSetMatrix(m, r, sizeof(double), A[i], lda, hdA[i], lda);
      hipblasSetMatrix(m, 1, sizeof(double), B[i], lda, hdB[i], lda);
    }
    hipMemcpy(dA, hdA, batchSize*sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(dB, hdB, batchSize*sizeof(double*), hipMemcpyHostToDevice);

    hipblasDgelsBatched(cublas_handle, HIPBLAS_OP_N, m, r, nrhs, dA, lda, dB, lda, &info, devInfoArray, batchSize);
    
    hipDeviceSynchronize();

    for(i=0; i<batchSize; i++) {
      hipblasGetMatrix(m, 1, sizeof(double), hdB[i], lda, B[i], lda);
    } 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    iterTime = 0;
    hipEventElapsedTime(&iterTime, start, stop);

    gpuTime = gpuTime + iterTime;
    double sse = 0; 
    for(i=0; i<batchSize; i++) {
      sse = 0;
      for(int j=r; j<m; j++) {
	sse = sse + B[i][j]*B[i][j];
      }
      if(sse < bestSSE[0]) {
	bestSSE[0] = sse;
	bestSubset[0] = r;
	bestSubset[1] = block*batchSize+i;
      }
    }
  }

  for(int i=0; i<batchSize; i++) {
    free(A[i]);
    free(B[i]);
    hipFree(hdA[i]);
    hipFree(hdB[i]);
  }
  if(dA) hipFree(dA);
  if(dB) hipFree(dB);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipblasDestroy(cublas_handle);
  if(A) free(A);
  if(B) free(B);
  if(hdA) free(hdA);
  if(hdB) free(hdB);

  return gpuTime;
}
  
int main(int argc, char **argv)
{

  // Make/read in a matrix/alamo file
  // Goal is to compare all subsets up to size ENUM using Givens Rotations

  // A simple example is ./batchSubset 10 10 2 0 

  int m = atoi(argv[1]);
  int n = atoi(argv[2]);
  int ENUM=atoi(argv[3]);
  int *bestSubset = (int *)malloc(2*sizeof(int));
  int *currentSubset = (int *)malloc(2*sizeof(int));
  double *bestSSE = (double *)malloc(sizeof(double));
  double currentSSE = 1e10;
  unsigned long ncr = 0;

  float batchTime = 0;
  float goldTime = 0;

  bestSSE[0] = currentSSE;
  bestSubset[0] = 0;
  bestSubset[1] = 0;
  // Generate subsets and calculate SSE, only store if optimal
  for(int r=100; r<=300; r+=50) {
    goldTime = testGold(m, 8, n, r);
    batchTime = testSubset(m, 8, n, r);
    printf("BatchSize=%d goldTPS=%f s gpuTPS=%f s\n", r, goldTime/r, batchTime/(1000*r));
  }
}